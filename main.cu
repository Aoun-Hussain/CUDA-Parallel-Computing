/*
Compilations Instructions: Use the following command to run:
                            "nvcc --std=c++11 *.cu -o executable.out"
*/

/*
 References:
    https://webpages.uncc.edu/abw/coit-grid01.uncc.edu/ITCS4145F12/Assignments/assign5F12.pdf
    http://www.joshiscorner.com/2013/12/2d-heat-conduction-solving-laplaces-equation-on-the-cpu-and-the-gpu/
*/

#include <iostream>
#include <fstream>
#include <cstdlib>
#include "unistd.h"
#include <list>
#include <cmath>
#include <algorithm>
#include <string>
#include "hip/hip_runtime.h"

#include <stdio.h>
using namespace std;

__global__ void Laplace(double *T_old, double *T_new, long X, long Y)
{
    /*
    GPU function to update the new array based on the values in the old array
    */

    //computing 2D indexes for a particular thread
    int i = blockIdx.x * blockDim.x + threadIdx.x ;
    int j = blockIdx.y * blockDim.y + threadIdx.y ;

    //computing 1D index from 2D indexes of point and its neighbors
    int P = i + j*X;
    int N = i + (j+1)*X;
    int S = i + (j-1)*X;
    int E = (i+1) + j*X;
    int W = (i-1) + j*X;

    //updating the interior node point
    if ((i > 0) && (i < (X-1)) && (j > 0) && (j < (Y-1)))
    {
        T_new[P] = 0.25*(T_old[E] + T_old[W] + T_old[N] + T_old[S]);
    }
}

void init(double *TEMP, long X, long Y)
{
    /*
    function to initialize the internal temperature of the plate
    and set boundary conditions
    */

    //setting all nodes of plate to 20 C (internal temp)
    for(int i = 0; i < X; i++)
    {
        for(int j = 0; j < Y; j++)
        {
            int idx = i + j*X;
            TEMP[idx] = 20.0000000000;
        }
    }

//
//    //setting Top of plate to 20 C
//    for(int i = 0; i < X; i++)
//    {
//        TEMP[i] = 20.000000;
//    }
//
//    //setting Bottom of plate to 20 C
//    for(int i = (Y-1)*X; i < (Y*X); i++)
//    {
//        TEMP[i] = 20.000000;
//    }
//
//    //setting Left of plate to 20 C
//    for(int j = 0; j < Y; j++)
//    {
//        int idx = j*X;
//        TEMP[idx] = 20.000000;
//    }
//
//    //setting Right of plate to 20 C
//    for(int j = 0; j < Y; j++)
//    {
//        int idx = j*X + (X-1);
//        TEMP[idx] = 20.000000;
//    }
//

    //setting 40% of top side to 100 C
    for(int i = 0; i < X; i++)
    {
        if ((i > 0.3*(X-1)) && (i < 0.7*(X-1)))
        {
            TEMP[i] = 100.0000000000;
        }
    }
}

bool isNumeric(const string &strIn, long &nInputNumber)
{
    /*
    Checks if the argument is numeric and returns true/false accordingly
    checks for the arguments -N and -I
    */

    bool bRC = all_of(strIn.begin(), strIn.end(), [](unsigned char c)
    {
        return ::isdigit(c);                      // http://www.cplusplus.com/reference/algorithm/all_of/
    }                                             // https://www.geeksforgeeks.org/lambda-expression-in-c/
    );                                            // http://www.cplusplus.com/reference/cctype/isdigit/

    if (bRC)
    {
        nInputNumber = stoul(strIn);              // https://www.cplusplus.com/reference/string/stoul/
        return true;
    }
    else
    {
        return false;
    }
}


int main(int argc, char* argv[])
{
    /*
    main function to check for all invalid combination of arguments -N and -I
    and performs multi-threaded laplace computations and write the coordinate values to csv file
    and outputs total computation time using cuda kernel
    */

    if (argc == 5)
    {
        long dim{ 0 };
        long iter{ 0 };
        string strInput1(argv[1]);    //-N
        string strInput2(argv[2]);    //positive integer
        string strInput3(argv[3]);    //-I
        string strInput4(argv[4]);    //positive integer

        if ((strInput1 != "-N") || (strInput3 != "-I"))
        {
            cout << "Invalid parameters, please check your values" << endl;
            return EXIT_SUCCESS;
        }

        bool bIsValid1 = isNumeric(strInput2, dim);
        bool bIsValid2 = isNumeric(strInput4, iter);

        if ((bIsValid1) && (bIsValid2) && (!strInput2.empty()) && (!strInput4.empty()) && (dim > 0) && (dim <= 256) && (iter >= 1) && (iter <= 10000))
        {
            long X = dim + 2; //assigning N+2 to X
            long Y = dim + 2; //assigning N+2 to Y
            long ITER = iter; //assigning iteration number

            double *T = new double[X*Y];   //allocating host memory
            double *_T1, *_T2;  //pointers to device (GPU) memory

            //initialize array on the host
            init(T, X, Y);

            //allocate storage space on the GPU
            hipMalloc((void **)&_T1,X*Y*sizeof(double));
            hipMalloc((void **)&_T2,X*Y*sizeof(double));

            //copy (initialized) host arrays to the GPU memory from CPU memory
            hipMemcpy(_T1, T, X*Y*sizeof(double), hipMemcpyHostToDevice);
            hipMemcpy(_T2, T, X*Y*sizeof(double), hipMemcpyHostToDevice);

            //assign a 2D distribution of CUDA "threads" within each CUDA "block"
            hipDeviceProp_t deviceProp;
            hipGetDeviceProperties(&deviceProp, 0);
            int ThreadsPerBlock = 16;//deviceProp.maxThreadsPerBlock;

            dim3 dimBlock(ThreadsPerBlock, ThreadsPerBlock);

            //calculate number of blocks along X and Y in a 2D CUDA "grid"
            dim3 dimGrid(ceil(double(X)/double(dimBlock.x)), ceil(double(Y)/double(dimBlock.y)), 1);

            float time;              //for measuring time
            hipEvent_t start, stop;

            hipEventCreate(&start);
            hipEventCreate(&stop);

            hipEventRecord(start);
            //begin Jacobi iteration
            int k = 0;
            while(k < ITER)
            {
                Laplace<<<dimGrid, dimBlock>>>(_T1, _T2, X, Y);   //update T1 using data stored in T2
                Laplace<<<dimGrid, dimBlock>>>(_T2, _T1, X, Y);   //update T2 using data stored in T1
                k += 2;
            }
            hipEventRecord(stop);

            //copy final array to the CPU from the GPU
            hipMemcpy(T, _T2, X*Y*sizeof(double), hipMemcpyDeviceToHost);
            hipDeviceSynchronize();

            hipEventSynchronize(stop);
            hipEventElapsedTime(&time, start, stop);

            float mul = pow(10.0, 2);         //rounding off to 2 decimal places
            float tm = ceil(time * mul) / mul;
            cout << tm << endl;

            //writing to csv file
            ofstream myfile ("finalTemperatures.csv");
            if (myfile.is_open())
            {
                for (int j = 0; j < Y; j++)
                {
                    string line;
                    for (int i = 0; i < X; i++)
                    {
                        int idx = i + j*X;

                        //rounding off to 10 decimal places
                        double multiplier = pow(10.0, 10);
                        double val = ceil(T[idx] * multiplier) / multiplier;
                        string elem = to_string(val) + ",";
                        line.append(elem);
                    }
                    myfile << line << "\n"; //comma separated values and new line after every row
                }
                myfile.close();
            }
            else
            {
                cout << "Unable to open file";
                return EXIT_SUCCESS;
            }

            // release memory on the host
            delete T;

            // release memory on the device
            hipFree(_T1);
            hipFree(_T2);

            //outputs execution time to console
            return 0;
        }
        else
        {
            cout << "Invalid parameters, please check your values" << endl;
            return EXIT_SUCCESS;
        }
    }
    else
    {
        cout << "Invalid parameters, please check your values" << endl;
        return EXIT_SUCCESS;
    }
}





